#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void addKernel(int* c, int* a, int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void addWithCuda(int* c, int* a, int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel <<<1, size >>> (dev_c, dev_a, dev_b);
    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}